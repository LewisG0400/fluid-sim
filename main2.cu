#include "hip/hip_runtime.h"
#include <iostream>

// Need to add this so the entry point is here instead of in SDL
#define SDL_MAIN_HANDLED
#include <GL/glew.h>
#include "SDL.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "utils/TexturedQuad.h"

void createTexturedQuad();

// Consider looking into uchar4 type
// https://stackoverflow.com/questions/26993351/is-there-a-penalty-to-using-char-variables-in-cuda-kernels
// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__SIMD.html#group__CUDA__MATH__INTRINSIC__SIMD
__global__ void render(unsigned int *pixels, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width || y >= height) return;
    int index = (x + y * width) * 3;
    pixels[0] = 1.0f;
    pixels[index+1] = 0.0f;
    pixels[index+2] = 1.0f;
}

int main(int agrc, char *argv[]) {
    int screenWidth = 800, screenHeight = 600;
    size_t imageSize = screenWidth * screenHeight * sizeof(float) * 3;
    bool running = true;

    hipError_t cErr;

    std::cout << "Done init" << std::endl;

    SDL_Init(SDL_INIT_VIDEO);

    SDL_Window *window = SDL_CreateWindow(
            "SDL2Test",
            SDL_WINDOWPOS_UNDEFINED,
            SDL_WINDOWPOS_UNDEFINED,
            screenWidth,
            screenHeight,
            SDL_WINDOW_OPENGL
    );

    SDL_GLContext context = SDL_GL_CreateContext(window);

    cErr = hipSetDevice(0);
    if(cErr != hipSuccess) std::cout << "Error Setting up device: " << hipGetErrorString(cErr) << std::endl;

    glewInit();
    glViewport(0, 0, screenWidth, screenHeight);
    glClearColor(1.0f, 0.0f, 0.0f, 1.0f);

    TexturedQuad q1 = TexturedQuad();

    // Generate a texture
    GLuint screenTexture;
    glGenTextures(1, &screenTexture);
    glBindTexture(GL_TEXTURE_2D, screenTexture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB, screenWidth, screenHeight, 0, GL_RGB, GL_FLOAT, NULL);
    q1.attachTexture(screenTexture);

    struct hipGraphicsResource* screenResourceHandle;

    cErr = hipGraphicsGLRegisterImage(&screenResourceHandle, screenTexture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);
    if(cErr != hipSuccess) std::cout << "Error registering the GL image with CUDA: " << hipGetErrorString(cErr) << std::endl;

    float *pixels;
    hipMalloc(&pixels, imageSize);
    hipMemset(pixels, 0, imageSize);

    float *hostPixels;
    hostPixels = (float*) malloc(imageSize);
    for(int i = 0; i < screenWidth * screenHeight * 3; i++) hostPixels[i] = 1.0f;
    hipMemcpy(pixels, hostPixels, screenWidth * screenHeight * 3, hipMemcpyHostToDevice);
    hipMemcpy(hostPixels, pixels, 100, hipMemcpyDeviceToHost);

    for(int i = 0; i < 100; i++) std::cout << (float) hostPixels[i] << std::endl;

    SDL_Event event;
    while(running) {
        SDL_PollEvent(&event);
        if(event.type == SDL_QUIT) {
            running = false;
        }

        dim3 blockSize(16, 16);
        dim3 gridSize((screenWidth + blockSize.x - 1) / blockSize.x, (screenHeight + blockSize.y - 1) / blockSize.y);

        //render<<<gridSize, blockSize>>>(pixels, screenWidth, screenHeight);

        hipDeviceSynchronize();

        hipArray *cudaArrayPointer;

        cErr = hipGraphicsMapResources(1, &screenResourceHandle, 0);
        if(cErr != hipSuccess) std::cout << "Error mapping the cuda resource: " << hipGetErrorString(cErr) << std::endl;

        cErr = hipGraphicsSubResourceGetMappedArray(&cudaArrayPointer, screenResourceHandle, 0, 0);
        if(cErr != hipSuccess) std::cout << "Error getting a mapped pointer for the resource: " << hipGetErrorString(cErr) << std::endl;

        hipMemcpyToArray(cudaArrayPointer, 0, 0, pixels, imageSize, hipMemcpyDeviceToDevice);

        hipGraphicsUnmapResources(1, &screenResourceHandle, 0);

        glClear(GL_COLOR_BUFFER_BIT);

        q1.render();

        SDL_GL_SwapWindow(window);
    }

    SDL_GL_DeleteContext(context);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}