#include <iostream>

// Need to add this so the entry point is here instead of in SDL
#define SDL_MAIN_HANDLED
#include <GL/glew.h>
#include "SDL.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "fluid/Fluid.cuh"

void createTexturedQuad();

int main(int agrc, char *argv[]) {
    int screenWidth = 1200, screenHeight = 960;
    bool running = true;

    hipError_t cErr;

    std::cout << "Done init" << std::endl;

    SDL_Init(SDL_INIT_VIDEO);

    SDL_Window *window = SDL_CreateWindow(
            "SDL2Test",
            SDL_WINDOWPOS_UNDEFINED,
            SDL_WINDOWPOS_UNDEFINED,
            screenWidth,
            screenHeight,
            SDL_WINDOW_OPENGL
    );

    SDL_GLContext context = SDL_GL_CreateContext(window);

    cErr = hipSetDevice(0);
    if(cErr != hipSuccess) std::cout << "Error Setting up device: " << hipGetErrorString(cErr) << std::endl;

    glewInit();

    glClearColor(1.0, 0.0, 0.0, 1.0);

    Fluid *fluid = new Fluid(screenWidth, screenHeight, 50, 50);

    SDL_Event event;
    while(running) {
        SDL_PollEvent(&event);
        if(event.type == SDL_QUIT) {
            running = false;
        }
        glClear(GL_COLOR_BUFFER_BIT);
        glDisable(GL_DEPTH_TEST);

        fluid->render();

        std::cout << "Frame done" << std::endl;

        SDL_GL_SwapWindow(window);
    }

    SDL_GL_DeleteContext(context);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}