#include "hip/hip_runtime.h"
#include <iostream>

// Need to add this so the entry point is here instead of in SDL
#define SDL_MAIN_HANDLED
#include <GL/glew.h>
#include "SDL.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "fluid/Fluid.h"

void createTexturedQuad();

// Consider looking into uchar4 type
// https://stackoverflow.com/questions/26993351/is-there-a-penalty-to-using-char-variables-in-cuda-kernels
// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__SIMD.html#group__CUDA__MATH__INTRINSIC__SIMD
__global__ void render(unsigned int *pixels, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int index = (x + y * width) * 3;
    pixels[index] = 255;
    pixels[index + 1] = 255;
    pixels[index + 2] = 0;
}

int main(int agrc, char *argv[]) {
    int screenWidth = 800, screenHeight = 600;
    size_t imageSize = screenWidth * screenHeight * 3 * sizeof(int);
    bool running = true;

    hipError_t cErr;

    std::cout << "Done init" << std::endl;

    SDL_Init(SDL_INIT_VIDEO);

    SDL_Window *window = SDL_CreateWindow(
            "Fluid",
            SDL_WINDOWPOS_UNDEFINED,
            SDL_WINDOWPOS_UNDEFINED,
            screenWidth,
            screenHeight,
            SDL_WINDOW_OPENGL
    );

    SDL_GLContext context = SDL_GL_CreateContext(window);

    cErr = hipSetDevice(0);
    if(cErr != hipSuccess) std::cout << "Error Setting up device: " << hipGetErrorString(cErr) << std::endl;

    glewInit();
    glViewport(0, 0, screenWidth, screenHeight);

    Fluid *fluid = new Fluid();

    SDL_Event event;
    while(running) {
        SDL_PollEvent(&event);
        if(event.type == SDL_QUIT) {
            running = false;
        }

        fluid->render();

        SDL_GL_SwapWindow(window);
    }

    SDL_GL_DeleteContext(context);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}