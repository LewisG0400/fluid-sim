#include "hip/hip_runtime.h"
#include <iostream>

// Need to add this so the entry point is here instead of in SDL
#define SDL_MAIN_HANDLED
#include <GL/glew.h>
#include "SDL.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include "utils/TexturedQuad.h"

void createTexturedQuad();

// Consider looking into uchar4 type
// https://stackoverflow.com/questions/26993351/is-there-a-penalty-to-using-char-variables-in-cuda-kernels
// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__SIMD.html#group__CUDA__MATH__INTRINSIC__SIMD
__global__ void render(unsigned int *pixels, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= width || y >= height) return;
    int index = (x + y * width) * 3;
    pixels[0] = 1.0f;
    pixels[index+1] = 0.0f;
    pixels[index+2] = 1.0f;
}

int main(int agrc, char *argv[]) {
    int screenWidth = 800, screenHeight = 600;
    size_t imageSize = screenWidth * screenHeight * sizeof(float) * 3;
    bool running = true;

    hipError_t cErr;

    std::cout << "Done init" << std::endl;

    SDL_Init(SDL_INIT_VIDEO);

    SDL_Window *window = SDL_CreateWindow(
            "SDL2Test",
            SDL_WINDOWPOS_UNDEFINED,
            SDL_WINDOWPOS_UNDEFINED,
            screenWidth,
            screenHeight,
            SDL_WINDOW_OPENGL
    );

    SDL_GLContext context = SDL_GL_CreateContext(window);

    cErr = hipSetDevice(0);
    if(cErr != hipSuccess) std::cout << "Error Setting up device: " << hipGetErrorString(cErr) << std::endl;

    glewInit();
    glViewport(0, 0, screenWidth, screenHeight);
    glClearColor(1.0f, 0.0f, 0.0f, 1.0f);

    struct hipGraphicsResource *frameBufferResourceHandle;

    cErr = hipGraphicsGLRegisterBuffer(&frameBufferResourceHandle, )

    SDL_Event event;
    while(running) {
        SDL_PollEvent(&event);
        if(event.type == SDL_QUIT) {
            running = false;
        }

        dim3 blockSize(16, 16);
        dim3 gridSize((screenWidth + blockSize.x - 1) / blockSize.x, (screenHeight + blockSize.y - 1) / blockSize.y);

        //render<<<gridSize, blockSize>>>(pixels, screenWidth, screenHeight);

        hipDeviceSynchronize();

        SDL_GL_SwapWindow(window);
    }

    SDL_GL_DeleteContext(context);
    SDL_DestroyWindow(window);
    SDL_Quit();

    return 0;
}