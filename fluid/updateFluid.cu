#include "hip/hip_runtime.h"
//
// Created by lewis on 25/05/2023.
//

#include "Particle.h"

#define SQRT_2_2 0.70710678118f

__device__ unsigned int getFieldIndex(unsigned int x, unsigned int y, unsigned int field_width);

__global__ void updateFluid(FieldCell *original_fluid, FieldCell *result_fluid, unsigned int fieldWidth, unsigned int fieldHeight) {
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= fieldWidth || y >= fieldHeight) return;

    unsigned int index = (x + y * fieldWidth);

    // Let result velocity be the divergence of density
    // Define indices for each neighbour
    unsigned int left_x = (x - 1) % 0xffff;
    unsigned int right_x = (x + 1) % fieldWidth;
    unsigned int down_y  = (y + 1) % fieldHeight;
    unsigned int up_y = (y - 1) % 0xffff;

    // Row by row for cache efficiency
    // Start with top row
    result_fluid[index].vx =
            // Top left
            -SQRT_2_2 * (original_fluid[getFieldIndex(left_x, up_y, fieldWidth)].density - original_fluid[index].density)
            // Top right
            + SQRT_2_2 * (original_fluid[getFieldIndex(right_x, up_y, fieldWidth)].density - original_fluid[index].density);
    result_fluid[index].vy =
            // Top left
            SQRT_2_2 * (original_fluid[getFieldIndex(left_x, up_y, fieldWidth)].density - original_fluid[index].density)
            // Top middle
            + (original_fluid[getFieldIndex(x, up_y, fieldWidth)].density - original_fluid[index].density)
            // Top right
            + SQRT_2_2 * (original_fluid[getFieldIndex(right_x, up_y, fieldWidth)].density - original_fluid[index].density);

    // Middle row
    result_fluid[index].vx +=
            // Middle left
            -(original_fluid[getFieldIndex(left_x, y, fieldWidth)].density - original_fluid[index].density)
            // Middle right
            + (original_fluid[getFieldIndex(right_x, y, fieldWidth)].density - original_fluid[index].density);

    // Bottom row
    result_fluid[index].vx +=
            // Bottom left
            -SQRT_2_2 * (original_fluid[getFieldIndex(left_x, down_y, fieldWidth)].density - original_fluid[index].density)
            // Bottom right
            + SQRT_2_2 * (original_fluid[getFieldIndex(right_x, down_y, fieldWidth)].density - original_fluid[index].density);
    result_fluid[index].vy -=
            // Top left
            SQRT_2_2 * (original_fluid[getFieldIndex(left_x, down_y, fieldWidth)].density - original_fluid[index].density)
            // Top middle
            + (original_fluid[getFieldIndex(x, down_y, fieldWidth)].density - original_fluid[index].density)
            // Top right
            + SQRT_2_2 * (original_fluid[getFieldIndex(right_x, down_y, fieldWidth)].density - original_fluid[index].density);

    // Gravity
    //result_fluid[index].vy -= 0.1f;

    // 0.125f to average over 8 neighbours
    result_fluid[index].density += result_fluid[index].vx * 0.125f;
    result_fluid[index].density += result_fluid[index].vy * 0.125f;

    if(pow(x - fieldWidth * 0.5, 2) + pow(y - fieldHeight * 0.5, 2) >= 3) result_fluid[index].density = 0.1f;

    if(result_fluid[index].density < 0.0f) result_fluid[index].density = 0.0f;
}