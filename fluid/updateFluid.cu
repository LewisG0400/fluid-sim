#include "hip/hip_runtime.h"
//
// Created by lewis on 25/05/2023.
//

#include "Particle.h"

__global__ void updateFluid(FieldCell *original_fluid, FieldCell *result_fluid, int fieldWidth, int fieldHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= fieldWidth || y >= fieldHeight) return;

    int index = (x + y * fieldWidth);

    for(int x_offset = -1; x_offset < 1; x_offset ++) {
        for(int y_offset = -1; y_offset < 1; y_offset ++) {
            if(x == 0 && y == 0) continue;
            int temp;
            temp = (x + x_offset) % fieldWidth;
            int cell_x = temp == -1 ? fieldWidth - 1 : temp;
            temp = (y + y_offset) % fieldHeight;
            int cell_y = temp == -1 ? fieldHeight - 1 : temp;

            int cell_index = (cell_x + cell_y * fieldWidth);

            result_fluid[index].vx += original_fluid[index].density - original_fluid[cell_index].density;
            result_fluid[index].vy += original_fluid[cell_index].vy;
        }
    }
}