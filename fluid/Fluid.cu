//
// Created by lewis on 23/05/2023.
//

#include "Fluid.h"

Fluid::Fluid() {
    Shader *metaballShader = new Shader("../resources/shaders/metaballShader");
    renderSurface = new RenderSurface();
    renderSurface->attachShader(metaballShader);

    hipError_t cErr;

    std::vector<float> startPositions;
    float startVelocities[100 * 2];
    for(int i = 0; i < N_PARTICLES; i+=2) {
        //startPositions.push_back((float) (i / 100.0f));
        //startPositions.push_back((float) (rand() % 100 - 50));
        startPositions.push_back(200.5f);
        startPositions.push_back(0.0f);
        startVelocities[i] = (rand() % 100 - 50) / 10.0f;
        startVelocities[i + 1] = (rand() % 100 - 50) / 10.0f;
    }

    // Setup the OpenGL position buffer
    glGenBuffers(1, &positionBuffer);
    glBindBuffer(GL_SHADER_STORAGE_BUFFER, positionBuffer);
    glBufferData(GL_SHADER_STORAGE_BUFFER, sizeof(startPositions), &startPositions[0], GL_STATIC_DRAW);
    glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 1, positionBuffer);

    // Attach the position buffer to a CUDA resource
    hipGraphicsGLRegisterBuffer(&positionResource, positionBuffer, hipGraphicsRegisterFlagsNone);

    glGenBuffers(1, &velocityBuffer);
    glBindBuffer(GL_SHADER_STORAGE_BUFFER, velocityBuffer);
    glBufferData(GL_SHADER_STORAGE_BUFFER, N_PARTICLES * sizeof(float) * 2, startVelocities, GL_STATIC_DRAW);
    glBindBufferBase(GL_SHADER_STORAGE_BUFFER, 2, velocityBuffer);
}

void Fluid::render() {
    renderSurface->render();
}