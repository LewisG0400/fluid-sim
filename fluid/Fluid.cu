#include "hip/hip_runtime.h"
//
// Created by lewis on 24/05/2023.
//

#include "Fluid.cuh"
#include <iostream>

// Consider looking into uchar4 type
// https://stackoverflow.com/questions/26993351/is-there-a-penalty-to-using-char-variables-in-cuda-kernels
// https://docs.nvidia.com/cuda/cuda-math-api/group__CUDA__MATH__INTRINSIC__SIMD.html#group__CUDA__MATH__INTRINSIC__SIMD
__global__ void setPixels(FieldCell *field, int fieldWidth, int fieldHeight, float *pixels, int screenWidth, int screenHeight) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if(x >= screenWidth || y >= screenHeight) return;

    int pixel_index = (x + y * screenWidth) * 3;
    int fieldIndex = ((int) floor(((float) fieldWidth / screenWidth) * x)) + ((int) floor(((float) fieldHeight / screenHeight) * y)) * fieldWidth;

    float density = field[fieldIndex].density;

    pixels[pixel_index] = density;
    pixels[pixel_index + 1] = 0.0f;
    pixels[pixel_index + 2] = density;
}

Fluid::Fluid(int screenWidth, int screenHeight, int fieldWidth, int fieldHeight) :
m_screenWidth(screenWidth), m_screenHeight(screenHeight), m_fieldWidth(fieldWidth), m_fieldHeight(fieldHeight) {
    hipError_t cErr;

    glViewport(0, 0, screenWidth, screenHeight);

    hipMalloc(&m_fluidField1, fieldWidth * fieldHeight * sizeof(FieldCell));
    hipMemset(m_fluidField1, 0, fieldWidth * fieldHeight * sizeof(FieldCell));

    hipMalloc(&m_fluidField2, fieldWidth * fieldHeight * sizeof(FieldCell));
    hipMemset(m_fluidField2, 0, fieldWidth * fieldHeight * sizeof(FieldCell));

    glGenBuffers(1, &m_pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, m_pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, screenWidth * screenHeight * sizeof(GLfloat) * 3, 0, GL_STREAM_DRAW);

    cErr = hipGraphicsGLRegisterBuffer(&m_cudaPboResource, m_pbo, cudaGraphicsMapFlagsWriteDiscard);
    if(cErr != hipSuccess) std::cout << "Error registering m_pbo with CUDA: " << hipGetErrorString(cErr) << std::endl;
}

void Fluid::render() {
    hipError_t cErr;

    FieldCell *old_fluid, *new_fluid;
    old_fluid = m_useBackBuffer == 0 ? m_fluidField1 : m_fluidField2;
    new_fluid = m_useBackBuffer == 0 ? m_fluidField2 : m_fluidField1;
    m_useBackBuffer ^= 1;

    dim3 blockSize1(1, 1);
    dim3 gridSize1(m_fieldWidth, m_fieldHeight);
    updateFluid<<<gridSize1, blockSize1>>>(old_fluid, new_fluid, m_fieldWidth, m_fieldHeight);
    cErr = hipGetLastError();
    if(cErr != hipSuccess) std::cout << "Error updating fluid:  " << hipGetErrorString(cErr) << std::endl;

    hipDeviceSynchronize();

    cErr = hipGraphicsMapResources(1, &m_cudaPboResource, 0);
    if(cErr != hipSuccess) std::cout << "Error mapping pbo: " << hipGetErrorString(cErr) << std::endl;

    cErr = hipGraphicsResourceGetMappedPointer((void**) &m_pixels, NULL, m_cudaPboResource);
    if(cErr != hipSuccess) std::cout << "Error getting mapped pointer to pbo: " << hipGetErrorString(cErr) << std::endl;

    dim3 blockSize(16, 16);
    dim3 gridSize((m_screenWidth + blockSize.x - 1) / blockSize.x, (m_screenHeight + blockSize.y - 1) / blockSize.y);
    setPixels<<<gridSize, blockSize>>>(new_fluid, m_fieldWidth, m_fieldHeight, m_pixels, m_screenWidth, m_screenHeight);
    cErr = hipGetLastError();
    if(cErr != hipSuccess) std::cout << "Error rendering:  " << hipGetErrorString(cErr) << std::endl;

    hipDeviceSynchronize();

    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, m_pbo);
    glRasterPos2i(-1, -1);
    glDrawPixels(m_screenWidth, m_screenHeight, GL_RGB, GL_FLOAT, 0);

    hipGraphicsUnmapResources(1, &m_cudaPboResource, 0);
}